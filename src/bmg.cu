/**
 * @file bmg.cu
 * @author AiglonDore
 * @brief Implements {@link bmg.h}
 * @version 0.1
 * @date 2023-12-06
 * 
 * @copyright Copyright (c) 2023
 * 
 */

#include "../header/bmg.h"
#include <random>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thread>

using namespace BMG;

void BMG::generate_default_BM(std::size_t n, std::vector<double>& BM, double start)
{
    BM.resize(n);
    BM[0] = start;

    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<double> d(0, 1);

    for (size_t i = 1; i < n; i++)
    {
        BM[i] = BM[i - 1] + d(gen);
    }

    BM.shrink_to_fit();
}

void BMG::generate_default_BM(std::size_t n, std::vector<std::vector<double>>& BM, std::size_t N, double start, bool cuda)
{
    BM.resize(N);

    if (cuda && N > std::thread::hardware_concurrency())
    {
        
    }
    else
    {
#ifdef OMP_ENABLED
        #pragma omp parallel for
        for (size_t i = 0; i < N; i++)
        {
            generate_default_BM(n, BM[i], start);
        }
#else
        std::thread *threads = new std::thread[N];
        for (size_t i = 0; i < N; i++)
        {
            threads[i] = std::thread([n, &BM, start, i]() {
                BMG::generate_default_BM(n, BM[i], start);
            });
        }
        for (size_t i = 0; i < N; i++)
        {
            threads[i].join();
        }

        delete[] threads;
        threads = nullptr;
#endif
    }
    
    BM.shrink_to_fit();
}