#include "hip/hip_runtime.h"
/**
 * @file bmg.cu
 * @author AiglonDore
 * @brief Implements {@link bmg.h}
 * @version 0.1
 * @date 2023-12-06
 * 
 * @copyright Copyright (c) 2023
 * 
 */

#include "../header/bmg.h"
#include <random>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thread>

using namespace BMG;

namespace std {
    size_t min(int a, size_t b)
    {
        if (a < 0)
            return 0;
        return (a < b) ? a : b;
    }
}

namespace CUDA{
    namespace Kernel {
        __global__ void generate_BM_kernel(double** BM, std::size_t n, double start, int offset)
        {
            int tid = blockIdx.x * blockDim.x + threadIdx.x;
            hiprandState state;

            hiprand_init(clock64(), tid + offset, 0, &state);
            
            BM[threadIdx.x + offset][0] = start;

            for (size_t i = 1; i < n; i++)
            {
                BM[threadIdx.x + offset][i] = BM[threadIdx.x + offset][i - 1] + hiprand_normal_double(&state);
            }
        }
    }

    void generate_default_BMs_on_cuda(std::size_t n, std::vector<std::vector<double>>& BM, std::size_t N, double start)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);

        int nb_max_thread = prop.maxThreadsPerBlock;

        double** d_BM;
        hipMalloc((void**)&d_BM, N * sizeof(double*));
        for (size_t i = 0; i < N; i++)
        {
            BM[i].resize(n);
            hipMalloc((void**)&d_BM[i], n * sizeof(double));
        }

        for (int i = 0; i < N / nb_max_thread + 1; i++)
            CUDA::Kernel::generate_BM_kernel<<<1, std::min(nb_max_thread, N)>>>(d_BM, n, start, i * nb_max_thread);

        for (size_t i = 0; i < N; i++)
        {
            hipMemcpy(BM[i].data(), d_BM[i], n * sizeof(double), hipMemcpyDeviceToHost);
            hipFree(d_BM[i]);
        }
        hipFree(d_BM);
        d_BM = nullptr;
    }
}

void BMG::generate_default_BM(std::size_t n, std::vector<double>& BM, double start)
{
    BM.resize(n);
    BM[0] = start;

    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<double> d(0, 1);

    for (size_t i = 1; i < n; i++)
    {
        BM[i] = BM[i - 1] + d(gen);
    }

    BM.shrink_to_fit();
}

void BMG::generate_default_BM(std::size_t n, std::vector<std::vector<double>>& BM, std::size_t N, double start, bool cuda)
{
    BM.resize(N);
    int nb_GPU(0);
    if (cuda && N > std::thread::hardware_concurrency() && hipGetDeviceCount(&nb_GPU) == hipSuccess && nb_GPU > 0)
    {
        CUDA::generate_default_BMs_on_cuda(n, BM, N, start);
    }
    else
    {
#ifdef _OPENMP
        #pragma omp parallel for
        for (size_t i = 0; i < N; i++)
        {
            generate_default_BM(n, BM[i], start);
        }
#else
        std::thread *threads = new std::thread[N];
        for (size_t i = 0; i < N; i++)
        {
            threads[i] = std::thread([n, &BM, start, i](void) -> void {
                BMG::generate_default_BM(n, BM[i], start);
            });
        }
        for (size_t i = 0; i < N; i++)
        {
            threads[i].join();
        }

        delete[] threads;
        threads = nullptr;
#endif
    }
    
    BM.shrink_to_fit();
}