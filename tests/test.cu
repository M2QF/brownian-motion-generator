/**
 * @file test.cu
 * @author AiglonDore
 * @brief Provides a test for the CUDA implementation of the algorithm
 * @version 1.0
 * @date 2023-12-06
 * 
 * @copyright Copyright (c) 2023
 * 
 */

#include "../header/bmg.h"
#include <CUnit/Basic.h>
#include <CUnit/CUnit.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstring>

using namespace std;

#define SIZE 1000
#define TOLERANCE 1.0 / SIZE * 100

/**
 * @brief Test the size of the vector
 * 
 */
void check_size()
{
    cout << "Testing the size of the vector" << endl;
    std::vector<double> BM;

    BMG::generate_default_BM(SIZE, BM);

    CU_ASSERT_EQUAL(BM.size(), SIZE);
}

void isNormalIncrement()
{
    cout << "Testing the normality of the increment" << endl;
    std::vector<double> BM;

    BMG::generate_default_BM(SIZE, BM);

    std::vector<double> increments;
    for (int i = 1; i < SIZE; i++)
    {
        increments.push_back(BM[i] - BM[i - 1]);
    }

    double mean = 0;
    for (int i = 0; i < SIZE - 1; i++)
    {
        mean += increments[i];
    }
    mean /= SIZE - 1;

    double variance = 0;
    for (int i = 0; i < SIZE - 1; i++)
    {
        variance += (increments[i] - mean) * (increments[i] - mean);
    }

    variance /= SIZE - 1;

    cout << "Mean : " << mean << endl;
    cout << "Variance : " << variance << endl;

    CU_ASSERT_DOUBLE_EQUAL(mean, 0, TOLERANCE);
    CU_ASSERT_DOUBLE_EQUAL(variance, 1, TOLERANCE);
}

/**
 * @brief Test the CUDA implementation of the algorithm
 * 
 * @param argc 
 * @param argv 
 * @return int 
 */
int main(int argc, char **argv)
{
    
    bool GPU = true;
    for (int i = 0; i < argc; i++)
    {
        if (strcmp(argv[i], "--cpu") == 0)
        {
            cout << "Assuming that the machine has no GPU" << endl;
            cout << "Testing the CPU implementation" << endl;
            GPU = false;
        }
    }

    CU_initialize_registry();

    CU_pSuite pSuite = CU_add_suite("Default Brownian motion", 0, 0);
    CU_ADD_TEST(pSuite, check_size);
    CU_ADD_TEST(pSuite, isNormalIncrement);

    CU_basic_set_mode(CU_BRM_VERBOSE);
    CU_basic_run_tests();

    CU_cleanup_registry();
    return 0;
}