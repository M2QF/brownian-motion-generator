/**
 * @file test.cu
 * @author AiglonDore
 * @brief Provides a test for the CUDA implementation of the algorithm
 * @version 1.0
 * @date 2023-12-06
 * 
 * @copyright Copyright (c) 2023
 * 
 */

#include "../header/bmg.h"
#include <CUnit/Basic.h>
#include <CUnit/CUnit.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstring>

using namespace std;

/**
 * @brief Test the CUDA implementation of the algorithm
 * 
 * @param argc 
 * @param argv 
 * @return int 
 */
int main(int argc, char **argv)
{
    CU_initialize_registry();
    bool GPU = true;
    for (int i = 0; i < argc; i++)
    {
        if (strcmp(argv[i], "--cpu") == 0)
        {
            cout << "Assuming that the machine has no GPU" << endl;
            cout << "Testing the CPU implementation" << endl;
            GPU = false;
        }
    }

    std::vector<double> BM;

    BMG::generate_default_BM(1000, BM);

    CU_ASSERT_EQUAL(BM.size(), 1000);

    CU_cleanup_registry();
    return 0;
}