/**
 * @file test.cu
 * @author AiglonDore
 * @brief Provides a test for the CUDA implementation of the algorithm
 * @version 1.0
 * @date 2023-12-06
 * 
 * @copyright Copyright (c) 2023
 * 
 */

#include "../header/bmg.h"
#include "dbmg.h"
#include <CUnit/Basic.h>
#include <CUnit/CUnit.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstring>

using namespace std;


/**
 * @brief Test the CUDA implementation of the algorithm
 * 
 * @param argc 
 * @param argv 
 * @return int 
 */
int main(int argc, char **argv)
{
    
    bool GPU = true;
    for (int i = 0; i < argc; i++)
    {
        if (strcmp(argv[i], "--cpu") == 0)
        {
            cout << "Assuming that the machine has no GPU" << endl;
            cout << "Testing the CPU implementation" << endl;
            GPU = false;
        }
    }

    CU_initialize_registry();

    CU_pSuite pSuite = CU_add_suite("Default Brownian motion", 0, 0);
    CU_ADD_TEST(pSuite, Tests::DBMG::check_size);
    CU_ADD_TEST(pSuite, Tests::DBMG::isNormalIncrement);

    CU_basic_set_mode(CU_BRM_VERBOSE);
    CU_basic_run_tests();

    CU_cleanup_registry();
    return 0;
}